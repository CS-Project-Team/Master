#include "hip/hip_runtime.h"
#include "gpuops.h"
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <sys/resource.h>
//#include <windows.h>

#define N 1000
#define INT_M 2
#define N_TESTS 1 
double get_time()
{
	struct timeval t;
	struct timezone tzp;
	gettimeofday(&t, &tzp);
	return t.tv_sec + t.tv_usec*1e-6;
}

const float CONST_FLOAT = 1.7;

__global__ void add_int(int *a, int *b, int *c, int n_blocks, int multiplier) {
	//int tid = blockDim.x*blockIdx.x + threadIdx.x;
	int tid = blockIdx.x;
	if(tid < n_blocks && tid < N){
		c[tid] = multiplier*a[tid] + b[tid];
//		c[tid] = c[tid] - 1;
//		b[tid] = a[tid] + c[tid];
//		c[tid] = c[tid] + b[tid];
//		a[tid] = a[tid] + c[tid];
//		c[tid] = c[tid] + a[tid];
	}
}

//TODO float operations
__global__ void add_float(float *a, float *b, float *c ) {
        int tid = blockIdx.x;
        if(tid < N){
                c[tid] = a[tid] + b[tid];
        }
}

//TODO read operations
__global__ void read_data() {
}

//TODO write operations
__global__ void write_data() {
}

void speed_test_int(int n_blocks, int n_threads){
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	hipMalloc( (void**)&dev_a, N * sizeof(int) );
	hipMalloc( (void**)&dev_b, N * sizeof(int) );
	hipMalloc( (void**)&dev_c, N * sizeof(int) );

	for( int i=0; i <N; i++) {
		a[i] = 2;
		b[i] = 1;
	}
	hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	
	//TODO clock_t start = clock(), diff;

	add_int<<<n_blocks,n_threads>>>(dev_a, dev_b, dev_c, n_blocks, INT_M);
	//add_int<<grids,blocks,1>>(dev_a, dev_b, dev_c);

	//TODO diff = clock() - start;
        double time = 0.0; //TODO (double) diff / (double) CLOCKS_PER_SEC;

	hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
	
	printf("\nTime spent: %d\n", time);
	for(int i=(N - 3); i<N; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i] );
	}

	hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_c );

}

void speed_test_float(int grids, int blocks){
        float a[N], b[N], c[N];
        float *dev_a, *dev_b, *dev_c;
        hipMalloc( (void**)&dev_a, N * sizeof(float) );
        hipMalloc( (void**)&dev_b, N * sizeof(float) );
        hipMalloc( (void**)&dev_c, N * sizeof(float) );

        for( int i=0; i <N; i++) {
                a[i] = CONST_FLOAT;
                b[i] = (float) i;
        }
        hipMemcpy( dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy( dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);

        //TODO clock_t start = clock(), diff;

        add_float<<<N,1>>>(dev_a, dev_b, dev_c);
        //add_float<<grids,blocks,1>>(dev_a, dev_b, dev_c);

        //TODO diff = clock() - start;
        double time = 0.0; //TODO (double) diff / (double) CLOCKS_PER_SEC;

        hipMemcpy( c, dev_c, N * sizeof(float), hipMemcpyDeviceToHost);

        printf("\nTime spent: %d\n", time);
        for(int i=(N - 3); i<N; i++) {
                printf("%f + %f = %f\n", a[i], b[i], c[i] );
        }

        hipFree( dev_a );
        hipFree( dev_b );
        hipFree( dev_c );
}

int ConvertSMVer2Cores(int major, int minor)
{
        // Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
        typedef struct {
                int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
                int Cores;
        } sSMtoCores;

        sSMtoCores nGpuArchCoresPerSM[] =
        { { 0x10,  8 }, // Tesla Generation (SM 1.0) G80 class
          { 0x11,  8 }, // Tesla Generation (SM 1.1) G8x class
          { 0x12,  8 }, // Tesla Generation (SM 1.2) G9x class
          { 0x13,  8 }, // Tesla Generation (SM 1.3) GT200 class
          { 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
          { 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
          { 0x30, 192}, // Fermi Generation (SM 3.0) GK10x class
          {   -1, -1 }
        };

        int index = 0;
        while (nGpuArchCoresPerSM[index].SM != -1) {
                if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor) ) {
                        return nGpuArchCoresPerSM[index].Cores;
                }
                index++;
        }
        printf("MapSMtoCores SM %d.%d is undefined (please update to the latest SDK)!\n", major, minor);
        return -1;
}


TestResult gpu_test() {
	TestResult result;
	double start, end;
	//number of blocks in a grid; number of threads in a block
	int n_blocks = 2, n_threads = 384; //384 get actual number of cores here
	
	int dev = 0;	
	hipSetDevice(0);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

	printf("\n  (%2d) Multiprocessors, (%3d) CUDA Cores/MP:     %d CUDA Cores\n",
               deviceProp.multiProcessorCount,
               ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
               ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);	
	for(int i = 0; i < N_TESTS; i++) {
		start = get_time();
		speed_test_int(n_blocks, n_threads);
		end = get_time();
		result.int_times[i] = end - start;
	}
	for(int i = 0; i < N_TESTS; i++) {
		//speed_test_float(n_grids, n_blocks);
                result.float_times[i] = 0;
        }

	//bandwidth test
	return result;
}

