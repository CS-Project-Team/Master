#include "hip/hip_runtime.h"
#include "gpuops.h"
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <sys/resource.h>
//#include <windows.h>

#define N 1000
#define INT_M 2
#define N_TESTS 1 
double get_time()
{
	struct timeval t;
	struct timezone tzp;
	gettimeofday(&t, &tzp);
	return t.tv_sec + t.tv_usec*1e-6;
}

const float CONST_FLOAT = 1.7;

__global__
void saxpy(int n, float a, float *x, float *y, float *z)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) {
	y[i] = a*x[i] + y[i];//y=4
	z[i] = a*x[i] + z[i];//z=5
	z[i] = a*y[i] + z[i];//z=13
	y[i] = a*x[i] + z[i];//y=15
        y[i] = a*x[i] + y[i];//y=17
	}
}

//TODO float operations
__global__ void add_float(float *a, float *b, float *c ) {
        int tid = blockIdx.x;
        if(tid < N){
                c[tid] = a[tid] + b[tid];
        }
}

//TODO read operations
__global__ void read_data() {
}

//TODO write operations
__global__ void write_data() {
}

void speed_test_int(int n_blocks, int n_threads){
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	hipMalloc( (void**)&dev_a, N * sizeof(int) );
	hipMalloc( (void**)&dev_b, N * sizeof(int) );
	hipMalloc( (void**)&dev_c, N * sizeof(int) );

	for( int i=0; i <N; i++) {
		a[i] = 2;
		b[i] = 1;
	}
	hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	
	//TODO clock_t start = clock(), diff;

	//add_int<<<n_blocks,n_threads>>>(dev_a, dev_b, dev_c, n_blocks, INT_M);
	//add_int<<grids,blocks,1>>(dev_a, dev_b, dev_c);

	//TODO diff = clock() - start;
        double time = 0.0; //TODO (double) diff / (double) CLOCKS_PER_SEC;

	hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
	
	printf("\nTime spent: %d\n", time);
	for(int i=(N - 3); i<N; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i] );
	}

	hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_c );

}

void speed_test_float(int grids, int blocks){
        float a[N], b[N], c[N];
        float *dev_a, *dev_b, *dev_c;
        hipMalloc( (void**)&dev_a, N * sizeof(float) );
        hipMalloc( (void**)&dev_b, N * sizeof(float) );
        hipMalloc( (void**)&dev_c, N * sizeof(float) );

        for( int i=0; i <N; i++) {
                a[i] = CONST_FLOAT;
                b[i] = (float) i;
        }
        hipMemcpy( dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy( dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);

        //TODO clock_t start = clock(), diff;

        add_float<<<N,1>>>(dev_a, dev_b, dev_c);
        //add_float<<grids,blocks,1>>(dev_a, dev_b, dev_c);

        //TODO diff = clock() - start;
        double time = 0.0; //TODO (double) diff / (double) CLOCKS_PER_SEC;

        hipMemcpy( c, dev_c, N * sizeof(float), hipMemcpyDeviceToHost);

        printf("\nTime spent: %d\n", time);
        for(int i=(N - 3); i<N; i++) {
                printf("%f + %f = %f\n", a[i], b[i], c[i] );
        }

        hipFree( dev_a );
        hipFree( dev_b );
        hipFree( dev_c );
}

int ConvertSMVer2Cores(int major, int minor)
{
        // Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
        typedef struct {
                int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
                int Cores;
        } sSMtoCores;

        sSMtoCores nGpuArchCoresPerSM[] =
        { { 0x10,  8 }, // Tesla Generation (SM 1.0) G80 class
          { 0x11,  8 }, // Tesla Generation (SM 1.1) G8x class
          { 0x12,  8 }, // Tesla Generation (SM 1.2) G9x class
          { 0x13,  8 }, // Tesla Generation (SM 1.3) GT200 class
          { 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
          { 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
          { 0x30, 192}, // Fermi Generation (SM 3.0) GK10x class
          {   -1, -1 }
        };

        int index = 0;
        while (nGpuArchCoresPerSM[index].SM != -1) {
                if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor) ) {
                        return nGpuArchCoresPerSM[index].Cores;
                }
                index++;
        }
        printf("MapSMtoCores SM %d.%d is undefined (please update to the latest SDK)!\n", major, minor);
        return -1;
}

TestResult gpu_test(){
  int Nz = 20 * (1 << 20);
  float *x, *y, *z, *d_x, *d_y, *d_z;
  x = (float*)malloc(Nz*sizeof(float));
  y = (float*)malloc(Nz*sizeof(float));
  z = (float*)malloc(Nz*sizeof(float));

  hipMalloc(&d_x, Nz*sizeof(float)); 
  hipMalloc(&d_y, Nz*sizeof(float));
  hipMalloc(&d_z, Nz*sizeof(float));

  for (int i = 0; i < Nz; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
    z[i] = 3.0f;
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipMemcpy(d_x, x, Nz*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, Nz*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_z, z, Nz*sizeof(float), hipMemcpyHostToDevice);

  hipEventRecord(start);

  printf("\nPerform SAXPY on %d elements\n", Nz);
  //saxpy<<<(Nz+511)/512, 512>>>(Nz, 2.0f, d_x, d_y);
  saxpy<<<(Nz+383)/384, 384>>>(Nz, 2.0f, d_x, d_y, d_z);

  hipEventRecord(stop);

  hipMemcpy(y, d_y, Nz*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(z, d_z, Nz*sizeof(float), hipMemcpyDeviceToHost);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  float maxError_y = 0.0f;
  float maxError_z = 0.0f;
  for (int i = 0; i < Nz; i++) {
    maxError_y = max(maxError_y, abs(y[i]-17.0f));
    maxError_z = max(maxError_z, abs(z[i]-13.0f));
  }

  printf("\nMax error y: %fn", maxError_y);
  printf("\nMax error z: %fn", maxError_z);
  printf("\nTime elapsed: %f", milliseconds/1e6);
  printf("\nEffective Bandwidth (GB/s): %f\n", Nz*4*3/milliseconds/1e6);

	TestResult result;
        for(int i = 0; i < N_TESTS; i++) {
                //speed_test_float(n_grids, n_blocks);
                result.float_times[i] = 0;
		result.int_times[i] = 0;
        }

        //bandwidth test
        return result;
}

TestResult gpu_test_old() {
	TestResult result;
	double start, end;
	//number of blocks in a grid; number of threads in a block
	int n_blocks = 2, n_threads = 384; //384 get actual number of cores here
	
	int dev = 0;	
	hipSetDevice(0);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

	printf("\n  (%2d) Multiprocessors, (%3d) CUDA Cores/MP:     %d CUDA Cores\n",
               deviceProp.multiProcessorCount,
               ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
               ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);	
	for(int i = 0; i < N_TESTS; i++) {
		start = get_time();
		speed_test_int(n_blocks, n_threads);
		end = get_time();
		result.int_times[i] = end - start;
	}
	for(int i = 0; i < N_TESTS; i++) {
		//speed_test_float(n_grids, n_blocks);
                result.float_times[i] = 0;
        }

	//bandwidth test
	return result;
}

