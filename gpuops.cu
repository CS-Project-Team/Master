#include "hip/hip_runtime.h"
#include "gpuops.h"
#include <stdio.h>
#include <time.h>

#define N 1000
#define N_TESTS 1 

const float CONST_FLOAT = 1.7;

__global__ void add_int(int *a, int *b, int *c ) {
	int tid = blockIdx.x;
	if(tid < N){
		c[tid] = a[tid] + b[tid];
	}
}

//TODO float operations
__global__ void add_float(float *a, float *b, float *c ) {
        int tid = blockIdx.x;
        if(tid < N){
                c[tid] = a[tid] + b[tid];
        }
}

//TODO read operations
__global__ void read_data() {
}

//TODO write operations
__global__ void write_data() {
}

void speed_test_int(int grids, int blocks){
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	hipMalloc( (void**)&dev_a, N * sizeof(int) );
	hipMalloc( (void**)&dev_b, N * sizeof(int) );
	hipMalloc( (void**)&dev_c, N * sizeof(int) );

	for( int i=0; i <N; i++) {
		a[i] = 1;
		b[i] = i;
	}
	hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	
	//TODO clock_t start = clock(), diff;

	add_int<<<N,1>>>(dev_a, dev_b, dev_c);
	//add_int<<grids,blocks,1>>(dev_a, dev_b, dev_c);

	//TODO diff = clock() - start;
        double time = 0.0; //TODO (double) diff / (double) CLOCKS_PER_SEC;

	hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
	
	printf("\nTime spent: %d\n", time);
	for(int i=(N - 3); i<N; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i] );
	}

	hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_c );

}

void speed_test_float(int grids, int blocks){
        float a[N], b[N], c[N];
        float *dev_a, *dev_b, *dev_c;
        hipMalloc( (void**)&dev_a, N * sizeof(float) );
        hipMalloc( (void**)&dev_b, N * sizeof(float) );
        hipMalloc( (void**)&dev_c, N * sizeof(float) );

        for( int i=0; i <N; i++) {
                a[i] = CONST_FLOAT;
                b[i] = (float) i;
        }
        hipMemcpy( dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy( dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);

        //TODO clock_t start = clock(), diff;

        add_float<<<N,1>>>(dev_a, dev_b, dev_c);
        //add_float<<grids,blocks,1>>(dev_a, dev_b, dev_c);

        //TODO diff = clock() - start;
        double time = 0.0; //TODO (double) diff / (double) CLOCKS_PER_SEC;

        hipMemcpy( c, dev_c, N * sizeof(float), hipMemcpyDeviceToHost);

        printf("\nTime spent: %d\n", time);
        for(int i=(N - 3); i<N; i++) {
                printf("%f + %f = %f\n", a[i], b[i], c[i] );
        }

        hipFree( dev_a );
        hipFree( dev_b );
        hipFree( dev_c );
}


TestResult gpu_test() {
	TestResult result;
	int n_blocks = 1, n_grids = 1; //get actual number of cores here
	
	for(int i = 0; i < N_TESTS; i++) {
		speed_test_int(n_grids, n_blocks);
		result.int_times[i] = 0;
	}
	for(int i = 0; i < N_TESTS; i++) {
		//speed_test_float(n_grids, n_blocks);
                result.float_times[i] = 0;
        }

	//bandwidth test
	return result;
}

